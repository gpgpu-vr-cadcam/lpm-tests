#include "hip/hip_runtime.h"
#include "RTreeMatcher.cuh"
#include <>

#include <thrust/scan.h>
#include <thrust/execution_policy.h>

__global__ void CopyMasks(int Count, int *R, int *rSums, int L, int** Masks, int *Lenghts, unsigned char *IPData)
{
	int mask = blockIdx.x * blockDim.x + threadIdx.x;

	while  (mask < Count)
	{
		int address = 0;
		Lenghts[mask] = IPData[mask * 5 + 4];

		int part;
		for (int i = 0; i < 4; ++i)
		{
			part = IPData[mask * 5 + i];
			address |= part << (8 * (3 - i));
		}

		for (int l = 0; l < L; ++l)
			Masks[l][mask] = (address >> (32 - rSums[l])) & ((2 << R[l] - 1) - 1);

		mask += blockDim.x * gridDim.x;
	}
}

__global__ void MarkNodesBorders(int Count, int l, int **nodesBorders, int **Masks)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
	while(i < Count)
	{
		if (Masks[l - 1][i - 1] != Masks[l - 1][i] || nodesBorders[l - 1][i] == 1)
			nodesBorders[l][i] = 1;

		i += blockDim.x * gridDim.x;
	}
}

__global__ void FillIndexes(int Count, int l, int **nodesIndexes, int **startIndexes, int **endIndexes)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
	while (i < Count)
	{
		if (nodesIndexes[l][i] > 0)
		{
			startIndexes[l][nodesIndexes[l][i] - 1] = i;
			endIndexes[l][nodesIndexes[l][i] - 2] = i;
		}

		i += blockDim.x * gridDim.x;
	}
}

__global__ void FillChildren(int l, int *LevelsSizes, int **startIndexes, int **endIndexes, int **Children, int *ChildrenCount, int **Masks, int **nodesIndexes)
{
	int node = blockIdx.x;
	while (node < LevelsSizes[l])
	{
		int i = startIndexes[l][node] + threadIdx.x;
		while (i < endIndexes[l][node])
		{
			if (nodesIndexes[l + 1][i] > 0)
				Children[l][node * ChildrenCount[l] + Masks[l][i]] = nodesIndexes[l + 1][i];

			i += blockDim.x;
		}
		node += gridDim.x;
	}
}

__global__ void FillListsLenghts(int l, int *R, int *rSums, int *rPreSums,  int *LevelsSizes, int **startIndexes, int **endIndexes, int *Lenghts, int **ListsLenghts)
{
	int node = blockIdx.x * blockDim.x + threadIdx.x;
	while(node < LevelsSizes[l])
	{
		int lenght = 0;
		for (int i = startIndexes[l][node]; i < endIndexes[l][node]; ++i)
			if (Lenghts[i] > rPreSums[l] && Lenghts[i] <= rSums[l])
				++lenght;

		ListsLenghts[l][node] = lenght;

		node += gridDim.x * blockDim.x;
	}

	//TODO: Ten kernel mo�na zrobi� lepiej. Bloki chodz� po w�z�ach na danym poziomie. W�tki zliczaj� sumy cz�ciowe. Potem redukcja w obr�bie bloku i wpisanie warto�ci.
}

__global__ void FillListItems(int l, int *R, int *rSums, int *rPreSums, int Count, int **startIndexes, int ** endIndexes, int **ListsStarts, int *LevelsSizes, int *Lenghts, int * ListItems)
{
	int node = blockIdx.x * blockDim.x + threadIdx.x;
	while(node < LevelsSizes[l])
	{
		int insertShift = 0;
		for (int maskLenght = rSums[l]; maskLenght > rPreSums[l]; --maskLenght)
		{
			for (int i = startIndexes[l][node]; i < endIndexes[l][node]; ++i)
				if (Lenghts[i] == maskLenght)
				{
					int index = (ListsStarts[l][node]) + insertShift;
					ListItems[index] = i;
					++insertShift;
				}
		}

		node += gridDim.x * blockDim.x;
	}

	//TODO: Ten kernel trzeba zrobi� lepiej, bo dla jednego w�z�a dzia�a tylko jednen w�tek. Prawdopodobnie trzeba b�dzie napisa� osobne kernele dla "ma�ych" i "du�ych" poziom�w i u�y� _atomicAdd na insertShift.
}

void RTreeModel::Build(IPSet set, GpuSetup setup)
{
	Count = set.Size;
	L = h_R.size();

	//Allocating memory for Rs
	GpuAssert(hipMalloc((void**)&R, L * sizeof(int)), "Cannot allocate memory for R");
	GpuAssert(hipMalloc((void**)&rSums, L * sizeof(int)), "Cannot allocate memory for R");
	GpuAssert(hipMalloc((void**)&rPreSums, L * sizeof(int)), "Cannot allocate memory for R");

	GpuAssert(hipMemcpy(R, h_R.data(), L * sizeof(int), hipMemcpyHostToDevice), "Cannot copy R memory");
	thrust::inclusive_scan(thrust::device, R, R + L, rSums);
	thrust::exclusive_scan(thrust::device, R, R + L, rPreSums);

	//TODO: Niepotrzebne jest budowanie w�z��w, je�eli �adna maska w zakresie nie jest dostatecznie d�uga.

	//Allocationg memory for masks
	GpuAssert(hipMalloc(reinterpret_cast<void**>(&Masks), L * sizeof(int*)), "Cannot init ip masks device memory");
	GpuAssert(hipMalloc(reinterpret_cast<void**>(&Lenghts), Count * sizeof(int)), "Cannot init Lenght mamory");

	int** h_Masks = new int*[L];
	for (int l = 0; l < L; ++l)
		GpuAssert(hipMalloc(reinterpret_cast<void**>(&h_Masks[l]), Count * sizeof(int)), "Cannot init ip masks device memory");
	GpuAssert(hipMemcpy(Masks, h_Masks, L * sizeof(int*), hipMemcpyHostToDevice), "Cannot copy Masks pointers to GPU");

	delete[] h_Masks;

	//Copying masks from IPSet and partitioning them
	CopyMasks <<< setup.Blocks, setup.Threads >>> (Count,  R, rSums,  L, Masks, Lenghts, set.d_IPData);
	GpuAssert(hipGetLastError(), "Error while launching CopyMasks kernel");
	GpuAssert(hipDeviceSynchronize(), "Error while running CopyMasks kernel");

	//Allocating memory for nodesBorders
	int ** nodesBorders;
	int ** nodesIndexes;
	GpuAssert(hipMalloc(reinterpret_cast<void**>(&nodesBorders), L * sizeof(int*)), "Cannot init nodes borders device memory");
	GpuAssert(hipMalloc(reinterpret_cast<void**>(&nodesIndexes), L * sizeof(int*)), "Cannot init nodes indexes device memory");

	int **h_nodesBorders = new int*[L];
	int **h_nodesIndexes = new int*[L];
	for(int l = 0; l < L; ++l)
	{
		GpuAssert(hipMalloc(reinterpret_cast<void**>(&h_nodesBorders[l]), Count * sizeof(int)), "Cannot init nodes borders device memory");
		GpuAssert(hipMalloc(reinterpret_cast<void**>(&h_nodesIndexes[l]), Count * sizeof(int)), "Cannot init nodes indexes device memory");
	}
	GpuAssert(hipMemcpy(nodesBorders, h_nodesBorders, L * sizeof(int*), hipMemcpyHostToDevice), "Cannot copy nodes borders device memory");
	GpuAssert(hipMemcpy(nodesIndexes, h_nodesIndexes, L * sizeof(int*), hipMemcpyHostToDevice), "Cannot copy nodes indexes device memory");

	//Marking first nodes on each level, setting rest of the nodesBorders memory to 0
	int mark = 1;
	for (int l = 0; l < L; ++l)
	{
		GpuAssert(hipMemset(h_nodesBorders[l], 0, Count * sizeof(int)), "Cannot clear nodesBorders memory");
		GpuAssert(hipMemcpy(h_nodesBorders[l], &mark, sizeof(int), hipMemcpyHostToDevice), "Cannot mark nodes start");
	}

	//Marking nodes borders
	for(int l = 1; l < L; ++l)
	{
		MarkNodesBorders <<<setup.Blocks, setup.Threads >>>(Count, l, nodesBorders, Masks);
		GpuAssert(hipGetLastError(), "Error while launching MarkNodesBorders kernel");
		GpuAssert(hipDeviceSynchronize(), "Error while running MarkNodesBorders kernel");
	}

	//Counting number of nodes and indexing them on each level. Indexing is done from 1 up, since 0 means empty value
	LevelsSizes = new int[L];

	for(int l = 0; l < L; ++l)
	{
		thrust::inclusive_scan(thrust::device, h_nodesBorders[l], h_nodesBorders[l] + Count, h_nodesIndexes[l]);
		GpuAssert(hipMemcpy(LevelsSizes + l, h_nodesIndexes[l] + Count - 1, sizeof(int), hipMemcpyDeviceToHost), "Cannot copy level size");
		thrust::transform(thrust::device, h_nodesBorders[l], h_nodesBorders[l] + Count, h_nodesIndexes[l], h_nodesIndexes[l], thrust::multiplies<int>());
	}

	int *d_LevelSizes;
	GpuAssert(hipMalloc((void**)&d_LevelSizes, L * sizeof(int)), "Cannot init d_LevelSizes memory");
	GpuAssert(hipMemcpy(d_LevelSizes, LevelsSizes, L * sizeof(int), hipMemcpyHostToDevice), "Cannot copy LevelSizes memory");

	//Filling start and end indexes of tree nodes
	int ** startIndexes;
	int ** endIndexes;
	GpuAssert(hipMalloc(reinterpret_cast<void**>(&startIndexes), L * sizeof(int*)), "Cannot init startIndexes device memory");
	GpuAssert(hipMalloc(reinterpret_cast<void**>(&endIndexes), L * sizeof(int*)), "Cannot init endIndexes device memory");

	int **h_startIndexes = new int*[L];
	int **h_endIndexes = new int*[L];
	for (int l = 0; l < L; ++l)
	{
		GpuAssert(hipMalloc(reinterpret_cast<void**>(&h_startIndexes[l]), LevelsSizes[l] * sizeof(int)), "Cannot init startIndexes device memory");
		GpuAssert(hipMalloc(reinterpret_cast<void**>(&h_endIndexes[l]), LevelsSizes[l] * sizeof(int)), "Cannot init endIndexes device memory");

		GpuAssert(hipMemset(h_startIndexes[l], 0, sizeof(int)), "Cannot mark first startIndex");
		GpuAssert(hipMemcpy(h_endIndexes[l] + (LevelsSizes[l] - 1), &Count, sizeof(int), hipMemcpyHostToDevice), "Cannot mark last endIndex");
	}
	GpuAssert(hipMemcpy(startIndexes, h_startIndexes, L * sizeof(int*), hipMemcpyHostToDevice), "Cannot copy startIndexes device memory");
	GpuAssert(hipMemcpy(endIndexes, h_endIndexes, L * sizeof(int*), hipMemcpyHostToDevice), "Cannot copy endIndexes device memory");

	for (int l = 1; l < L; ++l)
	{
		FillIndexes << <setup.Blocks, setup.Threads >> > (Count, l, nodesIndexes, startIndexes, endIndexes);
		GpuAssert(hipGetLastError(), "Error while launching FillIndexes kernel");
		GpuAssert(hipDeviceSynchronize(), "Error while running FillIndexes kernel");
	}

	//Filling children of tree nodes
	//TODO: Childrencount mog�oby by� L-1

	int *h_ChildrenCount = new int[L];
	for(int l = 0; l < L; ++l)
		h_ChildrenCount[l] = 2 << (h_R[l] - 1);

	GpuAssert(hipMalloc((void**)&ChildrenCount, L * sizeof(int)), "Cannot init Children memory");
	GpuAssert(hipMalloc((void**)&Children, (L-1) * sizeof(int*)), "Cannot init Children memory");

	GpuAssert(hipMemcpy(ChildrenCount, h_ChildrenCount, L * sizeof(int), hipMemcpyHostToDevice), "Cannot copy Children memory");

	h_Children = new int*[L];
	for(int l = 0; l < L-1; ++l)
		GpuAssert(hipMalloc((void**)&h_Children[l], LevelsSizes[l] * h_ChildrenCount[l] * sizeof(int)), "Cannot init children memory");

	GpuAssert(hipMemcpy(Children, h_Children, (L - 1) * sizeof(int*), hipMemcpyHostToDevice), "Cannot copy Children memory");

	delete[] h_ChildrenCount;

	for (int l = 0; l < L - 1; ++l)
	{
		FillChildren << <setup.Blocks, setup.Threads >> > (l, d_LevelSizes, startIndexes, endIndexes, Children, ChildrenCount, Masks, nodesIndexes);
		GpuAssert(hipGetLastError(), "Error while launching FillChildren kernel");
		GpuAssert(hipDeviceSynchronize(), "Error while running FillChildren kernel");
	}

	//Building lists of items for each node
	GpuAssert(hipMalloc((void**)&ListItems, Count * sizeof(int)), "Cannot init ListItems memory");
	GpuAssert(hipMalloc((void**)&ListsStarts, L * sizeof(int*)), "Cannot init ListsStarts memory");
	GpuAssert(hipMalloc((void**)&ListsLenghts, L * sizeof(int*)), "Cannot init ListsLenghts memory");

	h_ListsStarts = new int*[L];
	h_ListsLenghts = new int*[L];

	for(int l = 0; l < L; ++l)
	{
		GpuAssert(hipMalloc((void**)&h_ListsStarts[l], LevelsSizes[l] * sizeof(int)), "Cannot init ListsStarts memory");
		GpuAssert(hipMalloc((void**)&h_ListsLenghts[l], LevelsSizes[l] * sizeof(int)), "Cannot init ListsLenghts memory");
	}
	
	GpuAssert(hipMemcpy(ListsStarts, h_ListsStarts, L * sizeof(int*), hipMemcpyHostToDevice), "Cannot copy ListsStarts memory");
	GpuAssert(hipMemcpy(ListsLenghts, h_ListsLenghts, L * sizeof(int*), hipMemcpyHostToDevice), "Cannot copy ListsLenghts memory");

	for(int l = 0; l < L; ++l)
	{
		FillListsLenghts << <setup.Blocks, setup.Threads >> > (l, R, rSums, rPreSums, d_LevelSizes, startIndexes, endIndexes, Lenghts, ListsLenghts);
		GpuAssert(hipGetLastError(), "Error while launching FillListsLenghts kernel");
		GpuAssert(hipDeviceSynchronize(), "Error while running FillListsLenghts kernel");
	}

	//Filling lists start indexes
	int *totalListItemsPerLevel = new int[L];
	for(int l = 0; l < L; ++l)
	{
		thrust::exclusive_scan(thrust::device, h_ListsLenghts[l], h_ListsLenghts[l] + LevelsSizes[l], h_ListsStarts[l]);
		totalListItemsPerLevel[l] = thrust::reduce(thrust::device, h_ListsLenghts[l], h_ListsLenghts[l] + LevelsSizes[l]);
	}

	//Shifting lists
	int shift = 0;
	for (int l = 1; l < L; ++l)
	{
		shift += totalListItemsPerLevel[l - 1];
		thrust::for_each_n(thrust::device, h_ListsStarts[l], LevelsSizes[l], thrust::placeholders::_1 += shift);

		//TODO: Zerowanie start�w pustych list?
	}

	//Filling list items
	for(int l = 0; l < L; ++l)
	{
		//TODO: Ten kernel jest d�ugi, bo na poziomach gdzie jest ma�o w�z��w dzia�a ma�o w�tk�w
		FillListItems << <setup.Blocks, setup.Threads >> > (l, R, rSums, rPreSums, Count, startIndexes, endIndexes, ListsStarts, d_LevelSizes, Lenghts, ListItems);
		GpuAssert(hipGetLastError(), "Error while launching FillListItems kernel");
		GpuAssert(hipDeviceSynchronize(), "Error while running FillListItems kernel");
	}

	//Cleanup
	for(int i = 0; i < L; ++i)
	{
		GpuAssert(hipFree(h_nodesBorders[i]), "Cannot free nodes borders device memory.");
		GpuAssert(hipFree(h_nodesIndexes[i]), "Cannot free nodes indexes device memory.");
	}

	GpuAssert(hipFree(nodesBorders), "Cannot free nodes borders device memory.");
	GpuAssert(hipFree(nodesIndexes), "Cannot free nodes indexes device memory.");

	delete[] h_nodesBorders;
	delete[] h_nodesIndexes;

	GpuAssert(hipFree(d_LevelSizes), "Cannot free d_LevelSizes memory");

	for (int i = 0; i < L; ++i)
	{
		GpuAssert(hipFree(h_startIndexes[i]), "Cannot free startIndexes device memory.");
		GpuAssert(hipFree(h_endIndexes[i]), "Cannot free endIndexes device memory.");
	}

	GpuAssert(hipFree(startIndexes), "Cannot free startIndexes device memory.");
	GpuAssert(hipFree(endIndexes), "Cannot free endIndexes device memory.");

	delete[] h_startIndexes;
	delete[] h_endIndexes;

	delete[] totalListItemsPerLevel;
}

void RTreeModel::Dispose()
{
	if(Masks != NULL)
	{
		int** h_Masks = new int*[L];
		GpuAssert(hipMemcpy(h_Masks, Masks, L * sizeof(int*), hipMemcpyDeviceToHost), "Cannot copy Masks pointers to CPU");
		for (int i = 0; i < L; ++i)
			GpuAssert(hipFree(h_Masks[i]), "Cannot free Masks memory");
		delete[] h_Masks;
		GpuAssert(hipFree(Masks), "Cannot free Masks memory");
		Masks = NULL;
	}

	if(R != NULL)
	{
		GpuAssert(hipFree(R), "Cannot free R memory");
		GpuAssert(hipFree(rSums), "Cannot free rSums memory");
		GpuAssert(hipFree(rPreSums), "Cannot free rPreSums memory");

		R = rSums = rPreSums = NULL;
	}

	if(Lenghts != NULL)
	{
		GpuAssert(hipFree(Lenghts), "Cannot free Lenghts memory.");
		Lenghts = NULL;
	}

	if(LevelsSizes != NULL)
	{
		delete[] LevelsSizes;
		LevelsSizes = NULL;
	}

	if(Children != NULL)
	{
		for (int l = 0; l < L - 1; ++l)
			GpuAssert(hipFree(h_Children[l]), "Cannot free Children memory");

		GpuAssert(hipFree(Children), "Cannot free children memory");
		GpuAssert(hipFree(ChildrenCount), "Cannot free Children memory");
		delete[] h_Children;

		Children = h_Children = NULL;
		ChildrenCount = NULL;
	}

	if(ListItems != NULL)
	{
		GpuAssert(hipFree(ListItems), "Cannot free ListItems memory");
		ListItems = NULL;
	}

	if(ListsStarts != NULL)
	{
		for (int l = 0; l < L; ++l)
			GpuAssert(hipFree(h_ListsStarts[l]), "Cannot free ListsStarts memory");
		GpuAssert(hipFree(ListsStarts), "Cannot free ListsStarts memory");

		delete[] h_ListsStarts;
		ListsStarts = h_ListsStarts = NULL;
	}

	if (ListsLenghts != NULL)
	{
		for (int l = 0; l < L; ++l)
			GpuAssert(hipFree(h_ListsLenghts[l]), "Cannot free ListsLenghts memory");
		GpuAssert(hipFree(ListsLenghts), "Cannot free ListsLenghts memory");

		delete[] h_ListsLenghts;
		ListsLenghts = h_ListsLenghts = NULL;
	}
}

void RTreeResult::PrintResult()
{
}

int RTreeResult::CountMatched()
{
	int result = 0;

	for (int i = 0; i < IpsToMatchCount; ++i)
		if (MatchedMaskIndex[i] != -1)
			++result;
		else
			printf("%d\n", i);

	return result;
}

void RTreeMatcher::BuildModel(IPSet set)
{
	Setup = set.Setup;
	GpuAssert(hipSetDevice(Setup.DeviceID), "Cannot set cuda device in IPSet RandomSubset.");
	Timer timer;
	timer.Start();
	Model.Build(set, Setup);
	ModelBuildTime = timer.Stop();
	GpuAssert(hipSetDevice(0), "Cannot set cuda device in IPSet RandomSubset.");
}

__global__ void MatchIPs(int ** Children, int *ChildrenCount, int **Masks, int *result, int **ListsStarts, int **ListsLenghts, int *Lenghts, int L, int *R, int *rPreSums, int *ListItems,
	int **ips, int Count)
{
	extern __shared__ int m[];
	int *nodesToCheck = m + threadIdx.x * L;

	int i = blockDim.x * blockIdx.x + threadIdx.x;
	while( i < Count)
	{
		//Find nodes to be searched
		nodesToCheck[0] = 1;

		for (int l = 1; l < L; ++l)
		{
			nodesToCheck[l] = 0;
			if (nodesToCheck[l - 1] != 0)
				nodesToCheck[l] = Children[l - 1][(nodesToCheck[l - 1] - 1)*ChildrenCount[l - 1] + Masks[l - 1][i]];
			else
				break;
		}

		//Search lists
		for (int l = L - 1; l >= 0 && result[i] == -1; --l)
			if (nodesToCheck[l] != 0)
			{
				for (int s = ListsStarts[l][nodesToCheck[l] - 1];
					s < ListsStarts[l][nodesToCheck[l] - 1] + ListsLenghts[l][nodesToCheck[l] - 1] && result[i] == -1;
					++s)
				{
					int shitf = R[l] - (Lenghts[ListItems[s]] - rPreSums[l]);
					if (Masks[l][ListItems[s]] >> shitf == ips[l][i] >> shitf)
						result[i] = ListItems[s];
				}
			}

		i += gridDim.x * blockDim.x;
	}
}

RTreeResult RTreeMatcher::Match(IPSet set)
{
	RTreeResult result(set.Size);
	result.MatchedMaskIndex = new int[set.Size];

	Timer timer;
	timer.Start();

	int **d_IPs;
	int *d_IPsLenghts;

	GpuAssert(hipMalloc(reinterpret_cast<void**>(&d_IPs), Model.L * sizeof(int*)), "Cannot init ip masks device memory");
	GpuAssert(hipMalloc(reinterpret_cast<void**>(&d_IPsLenghts), set.Size * sizeof(int)), "Cannot init Lenght mamory");

	int** h_Masks = new int*[Model.L];
	for (int l = 0; l < Model.L; ++l)
		GpuAssert(hipMalloc(reinterpret_cast<void**>(&h_Masks[l]), Model.Count * sizeof(int)), "Cannot init ip masks device memory");
	GpuAssert(hipMemcpy(d_IPs, h_Masks, Model.L * sizeof(int*), hipMemcpyHostToDevice), "Cannot copy Masks pointers to GPU");

	delete[] h_Masks;

	//Copying ips from IPSet and partitioning them
	//TODO: Tutaj budowanie d_IPsLenghts jest niepotrzebne
	CopyMasks << < Setup.Blocks, Setup.Threads >> > (set.Size, Model.R, Model.rSums, Model.L, d_IPs, d_IPsLenghts, set.d_IPData);
	GpuAssert(hipGetLastError(), "Error while launching CopyMasks kernel");
	GpuAssert(hipDeviceSynchronize(), "Error while running CopyMasks kernel");

	int *d_Result;
	GpuAssert(hipMalloc((void**)&d_Result, result.IpsToMatchCount * sizeof(int)), "Cannot allocate memory for Result");
	thrust::fill_n(thrust::device, d_Result, result.IpsToMatchCount, -1);


	//Matching
	MatchIPs << <Setup.Blocks, Setup.Threads, Setup.Threads * Model.L * sizeof(int)>> > (Model.Children, Model.ChildrenCount, Model.Masks, d_Result, Model.ListsStarts, Model.ListsLenghts,
		Model.Lenghts, Model.L, Model.R, Model.rPreSums, Model.ListItems, d_IPs, set.Size);
	GpuAssert(hipGetLastError(), "Error while launching MatchIPs kernel");
	GpuAssert(hipDeviceSynchronize(), "Error while running MatchIPs kernel");

	GpuAssert(hipMemcpy(result.MatchedMaskIndex, d_Result, result.IpsToMatchCount * sizeof(int), hipMemcpyDeviceToHost), "Cannot copy Result data");

	GpuAssert(hipFree(d_Result), "Cannot free Result memory");
	GpuAssert(hipFree(d_IPs), "Cannot free d_IPs memory");
	GpuAssert(hipFree(d_IPsLenghts), "Cannot free d_IPsLenghts memory");

	result.MatchingTime = timer.Stop();

	return result;
}
