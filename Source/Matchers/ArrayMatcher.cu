#include "ArrayMatcher.cuh"
#include <thrust/execution_policy.h>

void ArrayMatcher::BuildModel(IPSet& set)
{
	Setup = set.Setup;
	GpuAssert(hipSetDevice(Setup.DeviceID), "Cannot set cuda device in IPSet RandomSubset.");
	Timer timer;
	timer.Start();

	//TODO: Implementacja

	ModelBuildTime = timer.Stop();
	GpuAssert(hipSetDevice(0), "Cannot set cuda device in IPSet RandomSubset.");
}

Result ArrayMatcher::Match(IPSet& set)
{
	Result result(set.Size);
	result.MatchedMaskIndex = new int[set.Size];

	Timer timer;
	timer.Start();

	int *d_Result;
	GpuAssert(hipMalloc((void**)&d_Result, result.IpsToMatchCount * sizeof(int)), "Cannot allocate memory for Result");
	thrust::fill_n(thrust::device, d_Result, result.IpsToMatchCount, -1);

	//TODO: Implementacja


	GpuAssert(hipMemcpy(result.MatchedMaskIndex, d_Result, result.IpsToMatchCount * sizeof(int), hipMemcpyDeviceToHost), "Cannot copy Result data");
	GpuAssert(hipFree(d_Result), "Cannot free Result memory");

	result.MatchingTime = timer.Stop();
	return result;
}